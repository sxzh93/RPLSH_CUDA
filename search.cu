#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <chrono>

#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hipblas.h>

// CUDA and CUBLAS functions
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>


#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h> // generate normal distribution

#include <thrust/sort.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/sequence.h>

#include "util.h"

#ifndef min
#define min(a,b) ((a < b) ? a : b)
#endif
#ifndef max
#define max(a,b) ((a > b) ? a : b)
#endif

using namespace std;

#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)


#define PROFILE 1

static std::chrono::_V2::system_clock::time_point ticker[6];
static double timer[6] = {0};
static const int ENCODE_QUERY_MATRIX = 0;
static const int COMPUTE_HAMMING_DISTANCE = 1;
static const int HAMMING_DISTANCE_SORTING = 2;
static const int COMPUTE_EUCLIDEAN_DISTANCE = 3;
static const int EUCLIDEAN_DISTANCE_SORTING = 4;
static const int GET_RESULT = 5;
#if PROFILE
#define START_ACTIVITY(X) ticker[X]=std::chrono::high_resolution_clock::now();
#define END_ACTIVITY(X) timer[X]+=(static_cast<std::chrono::duration<double>>(std::chrono::high_resolution_clock::now()-ticker[X])).count()
#define PRINT_PROFILER cout<<"encode query matrix:"<<timer[0]<<endl<<"compute hamming distance:"<<timer[1]<<endl<<"hamming distance sorting:"<<timer[2]<<endl<<"compute euclidean distance:"<<timer[3]<<endl<<"euclidean distance sorting:"<<timer[4]<<endl<<"get result:"<<timer[5]<<endl;
#else
#define START_ACTIVITY(X)
#define END_ACTIVITY(X)
#define PRINT_PROFILER
#endif


#define SQUARE4(v) v.x *= v.x; v.y *= v.y; v.z *= v.z; v.w *= v.w;
#define MINUS4(v1, v2) v1.x -= v2.x; v1.y -= v2.y; v1.z -= v2.z; v1.w -= v2.w;
#define REDUCE4(v) v.x += v.y; v.x += v.z; v.x += v.w;



__global__
void kernel_binarize(float* result_matrix, unsigned int* codes, int npoints, int ntables)
{
    int point_id = blockIdx.x * blockDim.x + threadIdx.x;
    int table_id = blockIdx.y * blockDim.y + threadIdx.y;
    if(point_id >= npoints || table_id >= ntables)
        return;

    unsigned int codelen = 32*ntables;
    unsigned int bit = 0;
    unsigned int result = 0;
    unsigned int column_start = table_id*32;
    for(int i=0;i<32;i++){
        if(result_matrix[point_id*codelen + column_start + i] > 0){
            bit = 1u << i;
            result |= bit;
        }
    }
    codes[table_id*npoints+point_id] = result;
}

__global__
void kernel_hamming_distance(unsigned int* d_query_codes, unsigned int* d_base_codes, unsigned int* d_hamming_distance, int nbase, int nquery, int ntable, unsigned int* d_hamming_distance_idx){
    unsigned int query_idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int base_idx = blockIdx.y * blockDim.y + threadIdx.y;

    if(base_idx>=nbase || query_idx>=nquery || query_idx*nbase + base_idx >= (nquery*nbase))
        return;

    int base_start = 0;
    int query_start = 0;
    unsigned int x;
    unsigned int result=0;
    for (size_t i = 0; i < ntable; i++) {
        x = d_base_codes[base_start+base_idx] ^ d_query_codes[query_start+query_idx];
        //compute number of 1 in x
        x = (x & 0x55555555) + ((x >> 1 ) & 0x55555555);
        x = (x & 0x33333333) + ((x >> 2 ) & 0x33333333);
        x = (x & 0x0f0f0f0f) + ((x >> 4 ) & 0x0f0f0f0f);
        x = (x & 0x00ff00ff) + ((x >> 8 ) & 0x00ff00ff);
        x = (x & 0x0000ffff) + ((x >> 16) & 0x0000ffff);

        result += x;
        base_start += nbase;
        query_start += nquery;
    }
    //d_hamming_distance[query_idx*nbase + base_idx] = result;
    d_hamming_distance[query_idx*nbase + base_idx] = result + (query_idx<<16);
    //d_hamming_distance[query_idx*nbase + base_idx] = result;
    d_hamming_distance_idx[query_idx*nbase + base_idx] = base_idx;
}



//TODO optimize IO
__global__
void kernel_euclidean_distance(float* d_query_matrix, float* d_base_matrix, unsigned int* d_hamming_distance_idx, double* d_euclidean_distance,  int nquery, int nbase, int L, int dim, float min_value, float max_value, unsigned int* d_euclidean_distance_idx){
    int query_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int L_idx = blockIdx.y * blockDim.y + threadIdx.y;

    if(query_idx>=nquery || L_idx>=L)
        return;

    int base_idx = d_hamming_distance_idx[query_idx*nbase + L_idx];
    double result = 0;
    int query_start = query_idx*dim;
    int base_start = base_idx*dim;
    double diff;
    for(int i=0;i<dim;i++){
        diff = d_query_matrix[query_start+i] - d_base_matrix[base_start+i];
        result += diff * diff;
    }
    //normalize result, so result is < 1
    result = result / (max_value*max_value*dim);
    d_euclidean_distance[query_idx*L + L_idx] = result + query_idx;
    d_euclidean_distance_idx[query_idx*L + L_idx] = base_idx;
}

__global__
void kernel_euclidean_distance_v2(float* d_query_matrix, float* d_base_matrix, unsigned int* d_hamming_distance_idx, double* d_euclidean_distance,  int nquery, int nbase, int L, int dim, float min_value, float max_value, unsigned int* d_euclidean_distance_idx){
    int query_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int L_idx = blockIdx.y * blockDim.y + threadIdx.y;

    if(query_idx>=nquery || L_idx>=L)
        return;

    int base_idx = d_hamming_distance_idx[query_idx*nbase + L_idx];
    int query_start = query_idx*dim/4;
    int base_start = base_idx*dim/4;

    float4 query_point;
    float4 base_point;
    double result = 0;
    for(int i=0;i<dim/4;i++){
        query_point = reinterpret_cast<float4*>(d_query_matrix)[query_start+i];
        base_point = reinterpret_cast<float4*>(d_base_matrix)[base_start+i];

        MINUS4(query_point, base_point);
        SQUARE4(query_point);
        REDUCE4(query_point);
        result += query_point.x;
    }
    //normalize result, so result is < 1
    result = result / (max_value*max_value*dim);
    d_euclidean_distance[query_idx*L + L_idx] = result + query_idx;
    d_euclidean_distance_idx[query_idx*L + L_idx] = base_idx;
}


__global__
void kernel_get_result(unsigned int* d_euclidean_distance_idx, unsigned int* d_result, int nquery, int K, int L){
    int query_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int K_idx = blockIdx.y * blockDim.y + threadIdx.y;

    if(query_idx>=nquery || K_idx>=K)
        return;

    d_result[query_idx*K + K_idx] = d_euclidean_distance_idx[query_idx*L+ K_idx];
}


void compute_index_gpu(float* d_A, float* d_B, unsigned int* d_codes, sMatrixSize &matrix_size, int npoint, int ntable){
    // alloc device memory to store projection result
    float *d_C;
    unsigned int size_C = matrix_size.uiWC * matrix_size.uiHC;
    unsigned int mem_size_C = sizeof(float) * size_C;
    checkCudaErrors(hipMalloc((void **) &d_C, mem_size_C));

    // setup execution parameters
    int block_size = 32;
    dim3 threads(block_size, block_size);
    dim3 grid(matrix_size.uiWC / threads.x, matrix_size.uiHC / threads.y);

    // execute the kernel, CUBLAS version 2.0
    const float alpha = 1.0f;
    const float beta  = 0.0f;
    hipblasHandle_t handle;
    checkCudaErrors(hipblasCreate(&handle));

    //matrix multiplication note cublas is column primary! need to transpose the order!
    checkCudaErrors(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, matrix_size.uiWB, matrix_size.uiHA, matrix_size.uiWA, &alpha, d_B, matrix_size.uiWB, d_A, matrix_size.uiWA, &beta, d_C, matrix_size.uiWB));

    dim3 threadsPerBlock(1024/ntable, ntable);
    dim3 numBlocks((npoint + threadsPerBlock.x -1) / threadsPerBlock.x, (ntable+threadsPerBlock.y-1) / threadsPerBlock.y);
    kernel_binarize<<<numBlocks, threadsPerBlock>>>(d_C, d_codes, npoint, ntable);
    checkCudaErrors(hipDeviceSynchronize());

    // clean up device memory
    checkCudaErrors(hipFree(d_C));

    // Destroy the handle
    checkCudaErrors(hipblasDestroy(handle));
}


void knn_prepare(float *&d_projection_matrix, unsigned int* &d_base_codes, float *&d_base_matrix, float *projection_matrix, unsigned int *base_codes, float *base_matrix, int dim, int ntable, int nbase){
    int codelen = ntable * 32;
    
    unsigned int size_projection_matrix = dim * codelen;
    unsigned int mem_size_projection_matrix = sizeof(float) * size_projection_matrix;
    unsigned int size_base_codes = nbase * ntable;
    unsigned int mem_size_base_codes = sizeof(unsigned int) * size_base_codes;
    unsigned int size_base_matrix = nbase * dim;
    unsigned int mem_size_base_matrix = sizeof(float) * size_base_matrix;

    checkCudaErrors(hipMalloc((void **) &d_projection_matrix, mem_size_projection_matrix));
    checkCudaErrors(hipMalloc((void **) &d_base_codes, mem_size_base_codes));
    checkCudaErrors(hipMalloc((void **) &d_base_matrix, mem_size_base_matrix));

    checkCudaErrors(hipMemcpy(d_projection_matrix, projection_matrix, mem_size_projection_matrix, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_base_codes, base_codes, mem_size_base_codes, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_base_matrix, base_matrix, mem_size_base_matrix, hipMemcpyHostToDevice));

}

void knn_free(float *&d_projection_matrix, unsigned int* &d_base_codes, float *&d_base_matrix){
    checkCudaErrors(hipFree(d_projection_matrix));
    checkCudaErrors(hipFree(d_base_codes));
    checkCudaErrors(hipFree(d_base_matrix));
}


void knn_search(unsigned int* result, float *d_base_matrix, float *query_matrix, float *d_projection_matrix, unsigned int* d_base_codes, unsigned int dim, unsigned int ntable, unsigned int nbase, unsigned int nquery, int L, int K, float min_value, float max_value){

    //device memory
    float *d_query_matrix;
    double *d_euclidean_distance;
    unsigned int *d_query_codes, *d_hamming_distance, *d_hamming_distance_idx, *d_euclidean_distance_idx, *d_result;
    int codelen = ntable * 32;

    // ============== Step1: Encodes query matrix =====================
    // this step will compute d_C = d_A * d_B and binarize d_C to d_codes

    // prepare d_projection_matrix and d_query code
    //Input: d_query_matrix
    START_ACTIVITY(ENCODE_QUERY_MATRIX);

    unsigned int size_query_matrix = nquery * dim;
    unsigned int mem_size_query_matrix = sizeof(float) * size_query_matrix;
    checkCudaErrors(hipMalloc((void **) &d_query_matrix, mem_size_query_matrix));
    checkCudaErrors(hipMemcpy(d_query_matrix, query_matrix, mem_size_query_matrix, hipMemcpyHostToDevice));

    //Output: d_query_codes
    unsigned int size_query_codes = nquery * ntable;
    unsigned int mem_size_query_codes = sizeof(unsigned int) * size_query_codes;
    checkCudaErrors(hipMalloc((void **) &d_query_codes, mem_size_query_codes));
    checkCudaErrors(hipMemset(d_query_codes, 0, mem_size_query_codes));

    sMatrixSize matrix_size;
    init_matrix_size(matrix_size, nquery, dim, codelen);
    compute_index_gpu(d_query_matrix, d_projection_matrix, d_query_codes, matrix_size, nquery, ntable);

    END_ACTIVITY(ENCODE_QUERY_MATRIX);

    //============== Step2: Compute Hamming Distance =====================
    //Input: d_query_codes, d_base_codes
    //Output: d_hamming_distance, d_hamming_distance_idx
    //Memory Cost: memcost = 2 * base_size * 1M (if nquery=1000, memcost=2G)

    //Input: d_base_codes
    START_ACTIVITY(COMPUTE_HAMMING_DISTANCE);

    //Output: hamming_distance
    unsigned int size_hamming_distance = nquery * nbase;
    unsigned int mem_hamming_distance  = sizeof(unsigned int) * size_hamming_distance;
    checkCudaErrors(hipMalloc((void **) &d_hamming_distance, mem_hamming_distance));

    //Onput: d_hamming_distance_idx
    unsigned int size_hamming_distance_idx = nquery * nbase;
    unsigned int mem_hamming_distance_idx  = sizeof(unsigned int) * size_hamming_distance_idx;
    checkCudaErrors(hipMalloc((void **) &d_hamming_distance_idx, mem_hamming_distance_idx));

    dim3 threadsPerBlock(1, 1024);
    dim3 numBlocks((nquery + threadsPerBlock.x -1) / threadsPerBlock.x, (nbase+threadsPerBlock.y-1) / threadsPerBlock.y);
    //kernel_hamming_distance_v1<<<numBlocks, threadsPerBlock>>>(d_query_codes, d_base_codes, d_hamming_distance, nbase, nquery, ntable);
    kernel_hamming_distance<<<numBlocks, threadsPerBlock>>>(d_query_codes, d_base_codes, d_hamming_distance, nbase, nquery, ntable, d_hamming_distance_idx);
    checkCudaErrors(hipDeviceSynchronize());

    //Useless: d_query_codes, d_base_codes,
    checkCudaErrors(hipFree(d_query_codes));

    END_ACTIVITY(COMPUTE_HAMMING_DISTANCE);


    //============== Step3: Sort According to Hamming Distance  =====================
    START_ACTIVITY(HAMMING_DISTANCE_SORTING);
    thrust::device_ptr<unsigned int> d_ptr_keys = thrust::device_pointer_cast(d_hamming_distance);
    thrust::device_ptr<unsigned int> d_ptr_values = thrust::device_pointer_cast(d_hamming_distance_idx);

    thrust::sort_by_key(d_ptr_keys, d_ptr_keys + size_hamming_distance, d_ptr_values);

    //Useless: d_hamming_distance
    checkCudaErrors(hipFree(d_hamming_distance));
    END_ACTIVITY(HAMMING_DISTANCE_SORTING);

    //============== Step4: Compute Euclidean Distance Bwtween Real Feature Vector =====================
    //Input: d_query_matrix, d_base_matrix, d_hamming_distance_idx
    //Output: d_euclidean_distance
    //Memory Cost: memcost = d_hamming_distance_idx + d_base_matrix = 1G + 1G, if nquery=1000, dim=960
    //Input: d_base_matrix

    START_ACTIVITY(COMPUTE_EUCLIDEAN_DISTANCE);

    //Output: d_euclidean_distance
    unsigned int size_euclidean_distance = nquery * L;
    unsigned int mem_size_euclidean_distance = sizeof(double) * size_euclidean_distance;
    checkCudaErrors(hipMalloc((void **) &d_euclidean_distance, mem_size_euclidean_distance));

    //Onput: d_euclidean_distance_idx
    unsigned int size_euclidean_distance_idx = nquery * L;
    unsigned int mem_size_euclidean_distance_idx  = sizeof(unsigned int) * size_euclidean_distance_idx;
    checkCudaErrors(hipMalloc((void **) &d_euclidean_distance_idx, mem_size_euclidean_distance_idx));

    threadsPerBlock.x = 1;
    threadsPerBlock.y = 1024;
    numBlocks.x = (nquery + threadsPerBlock.x -1) / threadsPerBlock.x;
    numBlocks.y = (L+threadsPerBlock.y-1) / threadsPerBlock.y;
    kernel_euclidean_distance_v2<<<numBlocks, threadsPerBlock>>>(d_query_matrix, d_base_matrix, d_hamming_distance_idx, d_euclidean_distance, nquery, nbase, L, dim, min_value, max_value, d_euclidean_distance_idx);
    checkCudaErrors(hipDeviceSynchronize());
    //Useless
    checkCudaErrors(hipFree(d_query_matrix));
    checkCudaErrors(hipFree(d_hamming_distance_idx));

    END_ACTIVITY(COMPUTE_EUCLIDEAN_DISTANCE);


    //============== Step5: Sort Base According to Euclidean Distance =====================
    START_ACTIVITY(EUCLIDEAN_DISTANCE_SORTING);
    thrust::device_ptr<double> d_ptr_keys_2 = thrust::device_pointer_cast(d_euclidean_distance);
    thrust::device_ptr<unsigned int> d_ptr_values_2 = thrust::device_pointer_cast(d_euclidean_distance_idx);
    thrust::sort_by_key(d_ptr_keys_2, d_ptr_keys_2 + size_euclidean_distance, d_ptr_values_2);

    //Useless: d_euclidean_distance
    checkCudaErrors(hipFree(d_euclidean_distance));

    END_ACTIVITY(EUCLIDEAN_DISTANCE_SORTING);

    //============== Step6: Get Result  =====================
    //Onput: d_result
    START_ACTIVITY(GET_RESULT);
    unsigned int size_result = nquery * K;
    unsigned int mem_size_result  = sizeof(unsigned int) * size_result;
    checkCudaErrors(hipMalloc((void **) &d_result, mem_size_result));

    threadsPerBlock.x = 32;
    threadsPerBlock.y = 32;
    numBlocks.x = (nquery + threadsPerBlock.x -1) / threadsPerBlock.x;
    numBlocks.y = (K+threadsPerBlock.y-1) / threadsPerBlock.y;
    kernel_get_result<<<numBlocks, threadsPerBlock>>>(d_euclidean_distance_idx, d_result, nquery, K, L);
    checkCudaErrors(hipDeviceSynchronize());

    //move result to CPU
    checkCudaErrors(hipMemcpy(result, d_result, mem_size_result, hipMemcpyDeviceToHost));

    //free memory
    checkCudaErrors(hipFree(d_result));
    checkCudaErrors(hipFree(d_euclidean_distance_idx));

    END_ACTIVITY(GET_RESULT);
}


int main(int argc, char** argv){

    float *base_matrix = NULL;
    float *query_matrix = NULL;
    float *projection_matrix = NULL;
    unsigned int *base_codes = NULL;
    unsigned int *result = NULL;

    unsigned int nbase=0;
    unsigned int nquery=0;
    unsigned int ntable=0;
    unsigned int base_dim=0;
    unsigned int query_dim=0;

    float *d_base_matrix, *d_projection_matrix;
    unsigned int * d_base_codes;

    // parse argument
    if(argc!=9){cout<< argv[0] << " index_file data_file query_file result_file ntable initsz querNN batch_size )" <<endl; exit(-1);}
    char* index_file = argv[1];
    char* base_file = argv[2];
    char* query_file = argv[3];
    char* result_file = argv[4];
    ntable = atoi(argv[5]);
    int L = atoi(argv[6]); // retrieve L points from index
    int K = atoi(argv[7]); // return K points at the end using real feature
    int batch_size = atoi(argv[8]);

    // load data and query
    load_data(base_file, base_matrix, nbase, base_dim);
    load_data(query_file, query_matrix, nquery, query_dim);

    // load index
    result = new unsigned int [nquery*K];
    load_index(index_file, base_codes, projection_matrix, base_dim, ntable, nbase);


    float min_value_query = *std::min_element(query_matrix, query_matrix + nquery*query_dim);
    float max_value_query = *std::max_element(query_matrix, query_matrix + nquery*query_dim);
    float min_value_base = *std::min_element(base_matrix, base_matrix + nbase * base_dim);
    float max_value_base = *std::max_element(base_matrix, base_matrix + nbase * base_dim);

    cout << "data load okay," << endl;
    std::cout << "min query " << min_value_query << '\n';
    std::cout << "max query " << max_value_query << '\n';
    std::cout << "min data " << min_value_base << '\n';
    std::cout << "max data " << max_value_base << '\n';

    // knn search
    cout << "begin knn search, batch size " << batch_size << endl;
    int n_remain = nquery;
    int n_completed = 0;
    auto s = std::chrono::high_resolution_clock::now();

    
    knn_prepare(d_projection_matrix, d_base_codes, d_base_matrix, projection_matrix, base_codes, base_matrix, base_dim, ntable, nbase);

    while(n_remain>0){
        int tmp_nquery = min(batch_size, n_remain);
        knn_search(result+n_completed*K, d_base_matrix, query_matrix + n_completed*query_dim, d_projection_matrix, d_base_codes, base_dim, ntable, nbase, tmp_nquery, L, K, min(min_value_base, min_value_query), max(max_value_base, max_value_query));
        n_remain -= tmp_nquery;
        n_completed += tmp_nquery;

    }
    auto e = std::chrono::high_resolution_clock::now();

    // report time
    std::chrono::duration<double> diff = e-s;
    std::cout << "query searching time: " << diff.count() << "\n";

    PRINT_PROFILER;
    saveKNNResults(result_file, result, nquery, K);


    knn_free(d_projection_matrix, d_base_codes, d_base_matrix);
    delete [] projection_matrix;
    delete [] base_codes;
    delete [] base_matrix;

    return 0;
}
